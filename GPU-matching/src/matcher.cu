#include "hip/hip_runtime.h"
#include "matcher.hpp"
#include "io.hpp"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_DIM 256

using namespace strum;

constexpr char CHUNK_SIZE = sizeof(chunk_t);
constexpr char NUM_NUCLEOTIDES = CHUNK_SIZE * io::Q;


__global__
void expand_kernel(byte_t* matrix, length_t length) {
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length) {
        auto current = matrix[idx];
        auto next = idx == length - 1? (byte_t) 0 : matrix[idx + 1];

        for (auto i = 2; i < CHAR_BIT; i += 2) {  // io::Q shifts of 2 bits each
            matrix[idx + i*length] = (byte_t) ((current << i) | (next >> (CHAR_BIT - i)));
        }
    }
}

void strum::copy_and_expand(const byte_t* bytes, byte_t* output, length_t length) {
    CUDA_CHECK(hipMalloc((void **) &output, length*io::Q))
    CUDA_CHECK(hipMemcpy(output, bytes, length, hipMemcpyHostToDevice))

    auto block_dim = BLOCK_DIM;
    auto grid_dim = length/block_dim + !!(length % block_dim);
    expand_kernel<<<grid_dim, block_dim>>>(output, length);
}

__device__ __forceinline__
byte_t hamming_distance(chunk_t x, chunk_t y) {
    auto diff = ~(x^y);
    diff &= (diff << 1);
    diff &= 0xAAAAAAAAAAAAAAAA;

    return (byte_t) (NUM_NUCLEOTIDES - __popcll(diff));
}

__global__
void min_hamming_distance_kernel(chunk_t sample, const byte_t* bytes, byte_t* result,
                                 length_t length, char excess = 0) {
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    auto limit = length - CHUNK_SIZE + 1 - !!excess;

    for (auto i = 0; i < io::Q; ++i) {
        if (idx*io::Q + i < limit*io::Q - excess) {
            auto chunk = (chunk_t) *(bytes + idx + i*length);
            auto dist = hamming_distance(sample, chunk);

            if (i == 0 || dist < result[idx]) {
                result[idx] = dist;
            }
        }
    }

    __syncthreads();

    auto stride = blockDim.x * gridDim.x;

    while (stride >>= 1) {
        if (idx < stride && idx + stride < limit) {
            result[idx] = min(result[idx], result[idx + stride]);
        }

        __syncthreads();
    }
}

char Matcher::min_hamming_distance(chunk_t sample) {
    byte_t* distances;

    CUDA_CHECK(hipMalloc((void**) &distances, length))

    auto block_dim = BLOCK_DIM;
    auto grid_dim = length/block_dim + !!(length % block_dim);

    min_hamming_distance_kernel<<<grid_dim, block_dim>>>(sample, d_bytes, distances, length, excess);

    char result = -1;

    CUDA_CHECK(hipMemcpy((byte_t*) &result, distances, 1, hipMemcpyDeviceToHost))
    CUDA_CHECK(hipFree(distances))

    return result;
}

Matcher::~Matcher() {
    CUDA_CHECK(hipFree(d_bytes))
}
